#include <hip/hip_runtime.h>

#define BLOCK_DIM 256


namespace
{
__device__ float DetMap2x(float sourcex, float sourcey, float ang) {
    return sourcex + sourcey * tan(ang);
}

__device__ float DetMap2y(float sourcex, float sourcey, float ang) {
    return sourcey + sourcex / tan(ang);
}

__device__ float Map2x(float sourcex, float sourcey, float pointx, float pointy) {
    return (sourcex * pointy - sourcey * pointx) / (pointy - sourcey);
}

__device__ float Map2y(float sourcex, float sourcey, float pointx, float pointy) {
    return (sourcey * pointx - sourcex * pointy) / (pointx - sourcex);
}

__device__ float CoordinateWeight(float sourcex, float sourcey, float pointx, float pointy) {
    return (sourcex - pointx) * (sourcex - pointx) + (sourcey - pointy) * (sourcey - pointy);
}

__device__ float TriAngCos(float a, float b, float c) {
    return abs(c) / sqrt(a * a + b * b + c * c);
}
}


__global__ void ProjConeArcDisCUDA3dKernel(
    float* __restrict__ Projection,
    hipTextureObject_t texObj,
    const float* __restrict__ ViewAngle,
    const int Width,
    const int Height,
    const int Depth,
    const int NumView,
    const int NumDetCol,
    const int NumDetRow,
    const float ImageSizeX,
    const float ImageSizeY,
    const float ImageSizeZ,
    const float DetColSize,
    const float DetRowSize,
    const float IsoSource,
    const float SourceDetector,
    const float PixXShift,
    const float PixYShift,
    const float PixZShift,
    const float BinColShift,
    const float BinRowShift) {

    const int idxBatch = blockIdx.x / NumView;
    const int idxView = blockIdx.x % NumView;
    const int idxDetRow = blockIdx.y;
    const int idxDetCol0 = blockIdx.z * blockDim.x;
    const int tx = threadIdx.x;
    const int idxDetCol = idxDetCol0 + tx;
    const int MaxNDetCol = ((idxDetCol0 + blockDim.x) > NumDetCol) ? (NumDetCol - idxDetCol0) : blockDim.x;
    __shared__ float ProjTemp[BLOCK_DIM];
    __shared__ float DetRow0Proj2Axis[BLOCK_DIM];
    __shared__ float DetRow1Proj2Axis[BLOCK_DIM];
    __shared__ float DetColProj2Axis[BLOCK_DIM + 1];

    float sinVal = sin(ViewAngle[idxView]);
    float cosVal = cos(ViewAngle[idxView]);
    float sourcex = - sinVal * IsoSource;
    float sourcey = cosVal * IsoSource;
    float DetRow0z = (idxDetRow - NumDetRow / 2.0) * DetRowSize + BinRowShift;
    float DetRow1z = DetRow0z + DetRowSize;
    float DetCol0Ang;
    float DetCol1Ang;
    int flag;
    if (cosVal * cosVal > 0.5) {
        ProjTemp[tx] = 0;
        float DetColy;
        if (cosVal >= 0) {
            DetCol0Ang = (idxDetCol0 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (idxDetCol + 1 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColy = - cos(DetCol1Ang - 0.5 * DetColSize) * SourceDetector + sourcey;
            flag = 1;
        } else {
            DetCol0Ang = (NumDetCol / 2.0 - idxDetCol0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (NumDetCol / 2.0 - idxDetCol - 1) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColy = - cos(DetCol1Ang + 0.5 * DetColSize) * SourceDetector + sourcey;
            flag = - 1;
        }
        float DetCol0Proj = DetMap2x(sourcex, sourcey, DetCol0Ang);
        float DetCol1Proj = DetMap2x(sourcex, sourcey, DetCol1Ang);
        float DetRow0Proj = sourcey / (sourcey - DetColy) * DetRow0z;
        float DetRow1Proj = sourcey / (sourcey - DetColy) * DetRow1z;
        if (tx == 0) DetColProj2Axis[tx] = DetCol0Proj;
        DetColProj2Axis[tx + 1] = DetCol1Proj;
        DetRow0Proj2Axis[tx] = DetRow0Proj;
        DetRow1Proj2Axis[tx] = DetRow1Proj;
        __syncthreads();
        float coef1 = (DetCol1Proj - DetColProj2Axis[tx]) * (DetRow1Proj - DetRow0Proj);
        float coef2 = TriAngCos((DetCol1Proj + DetColProj2Axis[tx]) / 2.0 - sourcex, (DetRow1Proj + DetRow0Proj) / 2.0, sourcey);
        float Point0x = - Width / 2.0 * ImageSizeX + PixXShift;
        float Point1x = Width / 2.0 * ImageSizeX + PixXShift;
        float Point0z = - Depth / 2.0 * ImageSizeZ + PixZShift;
        float Point1z = Depth / 2.0 * ImageSizeZ + PixZShift;
        for (int i = 0; (i * blockDim.x) < Height; i++) {
            int idxrow = i * blockDim.x + tx;
            if (idxrow < Height) {
                float Pointy = (Height / 2.0 - idxrow - 0.5) * ImageSizeY + PixYShift;
                float Point0Proj = Map2x(sourcex, sourcey, Point0x, Pointy);
                float Point1Proj = Map2x(sourcex, sourcey, Point1x, Pointy);
                float Point0Proj2z = sourcey / (sourcey - Pointy) * Point0z;
                float Point1Proj2z = sourcey / (sourcey - Pointy) * Point1z;
                float PixInterval = (Point1Proj - Point0Proj) / Width;
                float PixIntervalz = (Point1Proj2z - Point0Proj2z) / Depth;
                float tanVal0 = ((sourcex - Point0Proj) == 0)? 1e10 : sourcey / (sourcex - Point0Proj);
                float tanVal1 = ((BinColShift + ViewAngle[idxView]) == 0)? 1e10 : - 1 / tan(BinColShift + ViewAngle[idxView]);
                float delta = atan((tanVal0 - tanVal1) / (1 + tanVal0 * tanVal1));
                int idxd = floor(NumDetCol / 2.0 - idxDetCol0 + delta * flag / DetColSize);
                int idxcol;
                float Bound0;
                if (idxd < 0) {
                    Bound0 = DetColProj2Axis[0];
                    idxcol = floor((Bound0 - Point0Proj) / PixInterval);
                    idxd = 0;
                } else {
                    Bound0 = Point0Proj;
                    idxcol = 0;
                }
                Point1Proj = (idxcol + 1) * PixInterval + Point0Proj;
                if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                float temp = 0;
                while(idxcol < Width && idxd < MaxNDetCol) {
                    float Bound0z = max(Point0Proj2z, DetRow0Proj2Axis[idxd]);
                    float Bound1z = min(Point1Proj2z, DetRow1Proj2Axis[idxd]);
                    int idxslice;
                    if (Bound0z == Point0Proj2z) {
                        idxslice = 0;
                    } else {
                        idxslice = floor((Bound0z - Point0Proj2z) / PixIntervalz);
                    }
                    float Point1Proj2z = (idxslice + 1) * PixIntervalz + Point0Proj2z;
                    if (Point1Proj < DetCol1Proj) {
                        float coef = Point1Proj - Bound0;
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            temp += (Point1Proj2z - Bound0z) * coef * tex3D<float>(texObj, idxcol, idxrow, idxBatch * Depth + idxslice);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = Point1Proj;
                        idxcol++;
                        Point1Proj += PixInterval;
                    } else {
                        float coef = DetCol1Proj - Bound0;
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            temp += (Point1Proj2z - Bound0z) * coef * tex3D<float>(texObj, idxcol, idxrow, idxBatch * Depth + idxslice);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        atomicAdd(ProjTemp + idxd, temp);
                        temp = 0;
                        Bound0 = DetCol1Proj;
                        idxd ++;
                        if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                    }
                }
                if (temp != 0) atomicAdd(ProjTemp + idxd, temp);
            }
        }
        __syncthreads();
        if (idxDetCol < NumDetCol) {
            ProjTemp[tx] *= ImageSizeY / (coef1 * coef2);
            int idxDetColTemp = (cosVal >= 0) ? idxDetCol : (NumDetCol - 1 - idxDetCol);
            Projection[(blockIdx.x * NumDetRow + idxDetRow) * NumDetCol + idxDetColTemp] = ProjTemp[tx];
        }
    } else {
        ProjTemp[tx] = 0;
        float DetColx;
        if (sinVal >= 0) {
            DetCol0Ang = (idxDetCol0 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (idxDetCol + 1 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColx = sin(DetCol1Ang - 0.5 * DetColSize) * SourceDetector + sourcex;
            flag = 1;
        } else {
            DetCol0Ang = (NumDetCol / 2.0 - idxDetCol0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (NumDetCol / 2.0 - idxDetCol - 1) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColx = sin(DetCol1Ang + 0.5 * DetColSize) * SourceDetector + sourcex;
            flag = - 1;
        }
        float DetCol0Proj = DetMap2y(sourcex, sourcey, DetCol0Ang);
        float DetCol1Proj = DetMap2y(sourcex, sourcey, DetCol1Ang);
        float DetRow0Proj = sourcex / (sourcex - DetColx) * DetRow0z;
        float DetRow1Proj = sourcex / (sourcex - DetColx) * DetRow1z;
        if (tx == 0) DetColProj2Axis[tx] = DetCol0Proj;
        DetColProj2Axis[tx + 1] = DetCol1Proj;
        DetRow0Proj2Axis[tx] = DetRow0Proj;
        DetRow1Proj2Axis[tx] = DetRow1Proj;
        __syncthreads();
        float coef1 = (DetCol1Proj - DetColProj2Axis[tx]) * (DetRow1Proj - DetRow0Proj);
        float coef2 = TriAngCos((DetCol1Proj + DetColProj2Axis[tx]) / 2.0 - sourcey, (DetRow1Proj + DetRow0Proj) / 2.0, sourcex);
        float Point0y = - Height / 2.0 * ImageSizeY + PixYShift;
        float Point1y = Height / 2.0 * ImageSizeY + PixYShift;
        float Point0z = - Depth / 2.0 * ImageSizeZ + PixZShift;
        float Point1z = Depth / 2.0 * ImageSizeZ + PixZShift;
        for (int i = 0; (i * blockDim.x) < Width; i++) {
            int idxcol = i * blockDim.x + tx;
            if (idxcol < Width) {
                float Pointx = (idxcol - Width / 2.0 + 0.5) * ImageSizeX + PixXShift;
                float Point0Proj = Map2y(sourcex, sourcey, Pointx, Point0y);
                float Point1Proj = Map2y(sourcex, sourcey, Pointx, Point1y);
                float Point0Proj2z = sourcex / (sourcex - Pointx) * Point0z;
                float Point1Proj2z = sourcex / (sourcex - Pointx) * Point1z;
                float PixInterval = (Point1Proj - Point0Proj) / Height;
                float PixIntervalz = (Point1Proj2z - Point0Proj2z) / Depth;
                float tanVal0 = (sourcey - Point0Proj) / sourcex;
                float tanVal1 = - 1 / tan(BinColShift + ViewAngle[idxView]);
                float delta = atan((tanVal0 - tanVal1) / (1 + tanVal0 * tanVal1));
                int idxd = floor(NumDetCol / 2.0 - idxDetCol0 + delta * flag / DetColSize);
                int idxrow;
                float Bound0;
                if (idxd < 0) {
                    Bound0 = DetColProj2Axis[0];
                    idxrow = floor((Bound0 - Point0Proj) / PixInterval);
                    idxd = 0;
                } else {
                    Bound0 = Point0Proj;
                    idxrow = 0;
                }
                Point1Proj = (idxrow + 1) * PixInterval + Point0Proj;
                if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                float temp = 0;
                while(idxrow < Height && idxd < MaxNDetCol) {
                    float Bound0z = max(Point0Proj2z, DetRow0Proj2Axis[idxd]);
                    float Bound1z = min(Point1Proj2z, DetRow1Proj2Axis[idxd]);
                    int idxslice;
                    if (Bound0z == Point0Proj2z) {
                        idxslice = 0;
                    } else {
                        idxslice = floor((Bound0z - Point0Proj2z) / PixIntervalz);
                    }
                    float Point1Proj2z = (idxslice + 1) * PixIntervalz + Point0Proj2z;
                    if (Point1Proj < DetCol1Proj) {
                        float coef = Point1Proj - Bound0;
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            temp += (Point1Proj2z - Bound0z) * coef * tex3D<float>(texObj, idxcol, Height - 1 - idxrow, idxBatch * Depth + idxslice);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = Point1Proj;
                        idxrow++;
                        Point1Proj += PixInterval;
                    } else {
                        float coef = DetCol1Proj - Bound0;
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            temp += (Point1Proj2z - Bound0z) * coef * tex3D<float>(texObj, idxcol, Height - 1 - idxrow, idxBatch * Depth + idxslice);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        atomicAdd(ProjTemp + idxd, temp);
                        temp = 0;
                        Bound0 = DetCol1Proj;
                        idxd ++;
                        if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                    }
                }
                if (temp != 0) atomicAdd(ProjTemp + idxd, temp);
            }
        }
        __syncthreads();
        if (idxDetCol < NumDetCol) {
            ProjTemp[tx] *= ImageSizeX / (coef1 * coef2);
            int idxDetColTemp = (sinVal >= 0) ? idxDetCol : (NumDetCol - 1 - idxDetCol);
            Projection[(blockIdx.x * NumDetRow + idxDetRow) * NumDetCol + idxDetColTemp] = ProjTemp[tx];
        }
    }
}


__global__ void ProjTransConeArcDisCUDA3dKernel(
    float* __restrict__ Image,
    const float* __restrict__ Projection,
    const float* __restrict__ ViewAngle,
    const int Width,
    const int Height,
    const int Depth,
    const int NumView,
    const int NumDetCol,
    const int NumDetRow,
    const float ImageSizeX,
    const float ImageSizeY,
    const float ImageSizeZ,
    const float DetColSize,
    const float DetRowSize,
    const float IsoSource,
    const float SourceDetector,
    const float PixXShift,
    const float PixYShift,
    const float PixZShift,
    const float BinColShift,
    const float BinRowShift) {

    const int idxBatch = blockIdx.x / NumView;
    const int idxView = blockIdx.x % NumView;
    const int idxDetRow = blockIdx.y;
    const int idxDetCol0 = blockIdx.z * blockDim.x;
    const int tx = threadIdx.x;
    const int idxDetCol = idxDetCol0 + tx;
    const int MaxNDetCol = ((idxDetCol0 + blockDim.x) > NumDetCol) ? (NumDetCol - idxDetCol0) : blockDim.x;
    __shared__ float ProjTemp[BLOCK_DIM];
    __shared__ float DetRow0Proj2Axis[BLOCK_DIM];
    __shared__ float DetRow1Proj2Axis[BLOCK_DIM];
    __shared__ float DetColProj2Axis[BLOCK_DIM + 1];

    float sinVal = sin(ViewAngle[idxView]);
    float cosVal = cos(ViewAngle[idxView]);
    float sourcex = - sinVal * IsoSource;
    float sourcey = cosVal * IsoSource;
    float DetRow0z = (idxDetRow - NumDetRow / 2.0) * DetRowSize + BinRowShift;
    float DetRow1z = DetRow0z + DetRowSize;
    float DetCol0Ang;
    float DetCol1Ang;
    int flag;
    if (cosVal * cosVal > 0.5) {
        if (idxDetCol < NumDetCol) {
            int idxDetColTemp = (cosVal >= 0) ? idxDetCol : (NumDetCol - 1 - idxDetCol);
            ProjTemp[tx] = Projection[(blockIdx.x * NumDetRow + idxDetRow) * NumDetCol + idxDetColTemp];
        } else {
            ProjTemp[tx] = 0;
        }
        float DetColy;
        if (cosVal >= 0) {
            DetCol0Ang = (idxDetCol0 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (idxDetCol + 1 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColy = - cos(DetCol1Ang - 0.5 * DetColSize) * SourceDetector + sourcey;
            flag = 1;
        } else {
            DetCol0Ang = (NumDetCol / 2.0 - idxDetCol0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (NumDetCol / 2.0 - idxDetCol - 1) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColy = - cos(DetCol1Ang + 0.5 * DetColSize) * SourceDetector + sourcey;
            flag = - 1;
        }
        float DetCol0Proj = DetMap2x(sourcex, sourcey, DetCol0Ang);
        float DetCol1Proj = DetMap2x(sourcex, sourcey, DetCol1Ang);
        float DetRow0Proj = sourcey / (sourcey - DetColy) * DetRow0z;
        float DetRow1Proj = sourcey / (sourcey - DetColy) * DetRow1z;
        if (tx == 0) DetColProj2Axis[tx] = DetCol0Proj;
        DetColProj2Axis[tx + 1] = DetCol1Proj;
        DetRow0Proj2Axis[tx] = DetRow0Proj;
        DetRow1Proj2Axis[tx] = DetRow1Proj;
        __syncthreads();
        float coef1 = (DetCol1Proj - DetColProj2Axis[tx]) * (DetRow1Proj - DetRow0Proj);
        float coef2 = TriAngCos((DetCol1Proj + DetColProj2Axis[tx]) / 2.0 - sourcex, (DetRow1Proj + DetRow0Proj) / 2.0, sourcey);
        ProjTemp[tx] *= ImageSizeY / (coef1 * coef2);
        __syncthreads();
        float Point0x = - Width / 2.0 * ImageSizeX + PixXShift;
        float Point1x = Width / 2.0 * ImageSizeX + PixXShift;
        float Point0z = - Depth / 2.0 * ImageSizeZ + PixZShift;
        float Point1z = Depth / 2.0 * ImageSizeZ + PixZShift;
        for (int i = 0; (i * blockDim.x) < Height; i++) {
            int idxrow = i * blockDim.x + tx;
            if (idxrow < Height) {
                float Pointy = (Height / 2.0 - idxrow - 0.5) * ImageSizeY + PixYShift;
                float Point0Proj = Map2x(sourcex, sourcey, Point0x, Pointy);
                float Point1Proj = Map2x(sourcex, sourcey, Point1x, Pointy);
                float Point0Proj2z = sourcey / (sourcey - Pointy) * Point0z;
                float Point1Proj2z = sourcey / (sourcey - Pointy) * Point1z;
                float PixInterval = (Point1Proj - Point0Proj) / Width;
                float PixIntervalz = (Point1Proj2z - Point0Proj2z) / Depth;
                float tanVal0 = ((sourcex - Point0Proj) == 0)? 1e10 : sourcey / (sourcex - Point0Proj);
                float tanVal1 = ((BinColShift + ViewAngle[idxView]) == 0)? 1e10 : - 1 / tan(BinColShift + ViewAngle[idxView]);
                float delta = atan((tanVal0 - tanVal1) / (1 + tanVal0 * tanVal1));
                int idxd = floor(NumDetCol / 2.0 - idxDetCol0 + delta * flag / DetColSize);
                int idxcol;
                float Bound0;
                if (idxd < 0) {
                    Bound0 = DetColProj2Axis[0];
                    idxcol = floor((Bound0 - Point0Proj) / PixInterval);
                    idxd = 0;
                } else {
                    Bound0 = Point0Proj;
                    idxcol = 0;
                }
                Point1Proj = (idxcol + 1) * PixInterval + Point0Proj;
                if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                while(idxcol < Width && idxd < MaxNDetCol) {
                    float Bound0z = max(Point0Proj2z, DetRow0Proj2Axis[idxd]);
                    float Bound1z = min(Point1Proj2z, DetRow1Proj2Axis[idxd]);
                    int idxslice;
                    if (Bound0z == Point0Proj2z) {
                        idxslice = 0;
                    } else {
                        idxslice = floor((Bound0z - Point0Proj2z) / PixIntervalz);
                    }
                    float Point1Proj2z = (idxslice + 1) * PixIntervalz + Point0Proj2z;
                    if (Point1Proj < DetCol1Proj) {
                        float coef = Point1Proj - Bound0;
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            float temp = (Point1Proj2z - Bound0z) * coef * ProjTemp[idxd];
                            atomicAdd(Image + ((idxBatch * Depth + idxslice) * Height + idxrow) * Width + idxcol, temp);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = Point1Proj;
                        idxcol++;
                        Point1Proj += PixInterval;
                    } else {
                        float coef = DetCol1Proj - Bound0;
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            float temp = (Point1Proj2z - Bound0z) * coef * ProjTemp[idxd];
                            atomicAdd(Image + ((idxBatch * Depth + idxslice) * Height + idxrow) * Width + idxcol, temp);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = DetCol1Proj;
                        idxd ++;
                        if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                    }
                }
            }
        }
    } else {
        if (idxDetCol < NumDetCol) {
            int idxDetColTemp = (sinVal >= 0) ? idxDetCol : (NumDetCol - 1 - idxDetCol);
            ProjTemp[tx] = Projection[(blockIdx.x * NumDetRow + idxDetRow) * NumDetCol + idxDetColTemp];
        } else {
            ProjTemp[tx] = 0;
        }
        float DetColx;
        if (sinVal >= 0) {
            DetCol0Ang = (idxDetCol0 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (idxDetCol + 1 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColx = sin(DetCol1Ang - 0.5 * DetColSize) * SourceDetector + sourcex;
            flag = 1;
        } else {
            DetCol0Ang = (NumDetCol / 2.0 - idxDetCol0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (NumDetCol / 2.0 - idxDetCol - 1) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColx = sin(DetCol1Ang + 0.5 * DetColSize) * SourceDetector + sourcex;
            flag = - 1;
        }
        float DetCol0Proj = DetMap2y(sourcex, sourcey, DetCol0Ang);
        float DetCol1Proj = DetMap2y(sourcex, sourcey, DetCol1Ang);
        float DetRow0Proj = sourcex / (sourcex - DetColx) * DetRow0z;
        float DetRow1Proj = sourcex / (sourcex - DetColx) * DetRow1z;
        if (tx == 0) DetColProj2Axis[tx] = DetCol0Proj;
        DetColProj2Axis[tx + 1] = DetCol1Proj;
        DetRow0Proj2Axis[tx] = DetRow0Proj;
        DetRow1Proj2Axis[tx] = DetRow1Proj;
        __syncthreads();
        float coef1 = (DetCol1Proj - DetColProj2Axis[tx]) * (DetRow1Proj - DetRow0Proj);
        float coef2 = TriAngCos((DetCol1Proj + DetColProj2Axis[tx]) / 2.0 - sourcey, (DetRow1Proj + DetRow0Proj) / 2.0, sourcex);
        ProjTemp[tx] *= ImageSizeX / (coef1 * coef2);
        __syncthreads();
        float Point0y = - Height / 2.0 * ImageSizeY + PixYShift;
        float Point1y = Height / 2.0 * ImageSizeY + PixYShift;
        float Point0z = - Depth / 2.0 * ImageSizeZ + PixZShift;
        float Point1z = Depth / 2.0 * ImageSizeZ + PixZShift;
        for (int i = 0; (i * blockDim.x) < Width; i++) {
            int idxcol = i * blockDim.x + tx;
            if (idxcol < Width) {
                float Pointx = (idxcol - Width / 2.0 + 0.5) * ImageSizeX + PixXShift;
                float Point0Proj = Map2y(sourcex, sourcey, Pointx, Point0y);
                float Point1Proj = Map2y(sourcex, sourcey, Pointx, Point1y);
                float Point0Proj2z = sourcex / (sourcex - Pointx) * Point0z;
                float Point1Proj2z = sourcex / (sourcex - Pointx) * Point1z;
                float PixInterval = (Point1Proj - Point0Proj) / Height;
                float PixIntervalz = (Point1Proj2z - Point0Proj2z) / Depth;
                float tanVal0 = (sourcey - Point0Proj) / sourcex;
                float tanVal1 = - 1 / tan(BinColShift + ViewAngle[idxView]);
                float delta = atan((tanVal0 - tanVal1) / (1 + tanVal0 * tanVal1));
                int idxd = floor(NumDetCol / 2.0 - idxDetCol0 + delta * flag / DetColSize);
                int idxrow;
                float Bound0;
                if (idxd < 0) {
                    Bound0 = DetColProj2Axis[0];
                    idxrow = floor((Bound0 - Point0Proj) / PixInterval);
                    idxd = 0;
                } else {
                    Bound0 = Point0Proj;
                    idxrow = 0;
                }
                Point1Proj = (idxrow + 1) * PixInterval + Point0Proj;
                if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                while(idxrow < Height && idxd < MaxNDetCol) {
                    float Bound0z = max(Point0Proj2z, DetRow0Proj2Axis[idxd]);
                    float Bound1z = min(Point1Proj2z, DetRow1Proj2Axis[idxd]);
                    int idxslice;
                    if (Bound0z == Point0Proj2z) {
                        idxslice = 0;
                    } else {
                        idxslice = floor((Bound0z - Point0Proj2z) / PixIntervalz);
                    }
                    float Point1Proj2z = (idxslice + 1) * PixIntervalz + Point0Proj2z;
                    if (Point1Proj < DetCol1Proj) {
                        float coef = Point1Proj - Bound0;
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            float temp = (Point1Proj2z - Bound0z) * coef * ProjTemp[idxd];
                            atomicAdd(Image + ((idxBatch * Depth + idxslice) * Height + Height - 1 - idxrow) * Width + idxcol, temp);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = Point1Proj;
                        idxrow++;
                        Point1Proj += PixInterval;
                    } else {
                        float coef = DetCol1Proj - Bound0;
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            float temp = (Point1Proj2z - Bound0z) * coef * ProjTemp[idxd];
                            atomicAdd(Image + ((idxBatch * Depth + idxslice) * Height + Height - 1 - idxrow) * Width + idxcol, temp);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = DetCol1Proj;
                        idxd ++;
                        if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                    }
                }
            }
        }
    }
}


template<bool FBPWEIGHT>
__global__ void BackProjConeArcDisCUDA3dKernel(
    float* __restrict__ Image,
    const float* __restrict__ Projection,
    const float* __restrict__ ViewAngle,
    const int Width,
    const int Height,
    const int Depth,
    const int NumView,
    const int NumDetCol,
    const int NumDetRow,
    const float ImageSizeX,
    const float ImageSizeY,
    const float ImageSizeZ,
    const float DetColSize,
    const float DetRowSize,
    const float IsoSource,
    const float SourceDetector,
    const float PixXShift,
    const float PixYShift,
    const float PixZShift,
    const float BinColShift,
    const float BinRowShift) {

    const int idxBatch = blockIdx.x / NumView;
    const int idxView = blockIdx.x % NumView;
    const int idxDetRow = blockIdx.y;
    const int idxDetCol0 = blockIdx.z * blockDim.x;
    const int tx = threadIdx.x;
    const int idxDetCol = idxDetCol0 + tx;
    const int MaxNDetCol = ((idxDetCol0 + blockDim.x) > NumDetCol) ? (NumDetCol - idxDetCol0) : blockDim.x;
    __shared__ float ProjTemp[BLOCK_DIM];
    __shared__ float DetRow0Proj2Axis[BLOCK_DIM];
    __shared__ float DetRow1Proj2Axis[BLOCK_DIM];
    __shared__ float DetColProj2Axis[BLOCK_DIM + 1];

    float sinVal = sin(ViewAngle[idxView]);
    float cosVal = cos(ViewAngle[idxView]);
    float sourcex = - sinVal * IsoSource;
    float sourcey = cosVal * IsoSource;
    float DetColRad = IsoSource * DetColSize;
    float DetRow0z = (idxDetRow - NumDetRow / 2.0) * DetRowSize + BinRowShift;
    float DetRow1z = DetRow0z + DetRowSize;
    float DetCol0Ang;
    float DetCol1Ang;
    int flag;
    if (cosVal * cosVal > 0.5) {
        if (idxDetCol < NumDetCol) {
            int idxDetColTemp = (cosVal >= 0) ? idxDetCol : (NumDetCol - 1 - idxDetCol);
            ProjTemp[tx] = Projection[(blockIdx.x * NumDetRow + idxDetRow) * NumDetCol + idxDetColTemp] * DetColRad;
        } else {
            ProjTemp[tx] = 0;
        }
        float DetColy;
        if (cosVal >= 0) {
            DetCol0Ang = (idxDetCol0 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (idxDetCol + 1 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColy = - cos(DetCol1Ang - 0.5 * DetColSize) * SourceDetector + sourcey;
            flag = 1;
        } else {
            DetCol0Ang = (NumDetCol / 2.0 - idxDetCol0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (NumDetCol / 2.0 - idxDetCol - 1) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColy = - cos(DetCol1Ang + 0.5 * DetColSize) * SourceDetector + sourcey;
            flag = - 1;
        }
        float DetCol0Proj = DetMap2x(sourcex, sourcey, DetCol0Ang);
        float DetCol1Proj = DetMap2x(sourcex, sourcey, DetCol1Ang);
        float DetRow0Proj = sourcey / (sourcey - DetColy) * DetRow0z;
        float DetRow1Proj = sourcey / (sourcey - DetColy) * DetRow1z;
        if (tx == 0) DetColProj2Axis[tx] = DetCol0Proj;
        DetColProj2Axis[tx + 1] = DetCol1Proj;
        DetRow0Proj2Axis[tx] = DetRow0Proj;
        DetRow1Proj2Axis[tx] = DetRow1Proj;
        __syncthreads();
        float Point0x = - Width / 2.0 * ImageSizeX + PixXShift;
        float Point1x = Width / 2.0 * ImageSizeX + PixXShift;
        float Point0z = - Depth / 2.0 * ImageSizeZ + PixZShift;
        float Point1z = Depth / 2.0 * ImageSizeZ + PixZShift;
        for (int i = 0; (i * blockDim.x) < Height; i++) {
            int idxrow = i * blockDim.x + tx;
            if (idxrow < Height) {
                float Pointy = (Height / 2.0 - idxrow - 0.5) * ImageSizeY + PixYShift;
                float Point0Proj = Map2x(sourcex, sourcey, Point0x, Pointy);
                float Point1Proj = Map2x(sourcex, sourcey, Point1x, Pointy);
                float Point0Proj2z = sourcey / (sourcey - Pointy) * Point0z;
                float Point1Proj2z = sourcey / (sourcey - Pointy) * Point1z;
                float PixInterval = (Point1Proj - Point0Proj) / Width;
                float PixIntervalz = (Point1Proj2z - Point0Proj2z) / Depth;
                float PixArea = PixInterval * PixIntervalz;
                float tanVal0 = ((sourcex - Point0Proj) == 0)? 1e10 : sourcey / (sourcex - Point0Proj);
                float tanVal1 = ((BinColShift + ViewAngle[idxView]) == 0)? 1e10 : - 1 / tan(BinColShift + ViewAngle[idxView]);
                float delta = atan((tanVal0 - tanVal1) / (1 + tanVal0 * tanVal1));
                int idxd = floor(NumDetCol / 2.0 - idxDetCol0 + delta * flag / DetColSize);
                int idxcol;
                float Bound0;
                if (idxd < 0) {
                    Bound0 = DetColProj2Axis[0];
                    idxcol = floor((Bound0 - Point0Proj) / PixInterval);
                    idxd = 0;
                } else {
                    Bound0 = Point0Proj;
                    idxcol = 0;
                }
                float Pointx = (idxcol + 0.5) * ImageSizeX + Point0x;
                Point1Proj = (idxcol + 1) * PixInterval + Point0Proj;
                if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                while(idxcol < Width && idxd < MaxNDetCol) {
                    float Bound0z = max(Point0Proj2z, DetRow0Proj2Axis[idxd]);
                    float Bound1z = min(Point1Proj2z, DetRow1Proj2Axis[idxd]);
                    int idxslice;
                    if (Bound0z == Point0Proj2z) {
                        idxslice = 0;
                    } else {
                        idxslice = floor((Bound0z - Point0Proj2z) / PixIntervalz);
                    }
                    float Point1Proj2z = (idxslice + 1) * PixIntervalz + Point0Proj2z;
                    if (Point1Proj < DetCol1Proj) {
                        float coef = (Point1Proj - Bound0) / PixArea;
                        if (FBPWEIGHT) coef /= CoordinateWeight(sourcex, sourcey, Pointx, Pointy);
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            float temp = (Point1Proj2z - Bound0z) * coef * ProjTemp[idxd];
                            atomicAdd(Image + ((idxBatch * Depth + idxslice) * Height + idxrow) * Width + idxcol, temp);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = Point1Proj;
                        idxcol++;
                        Pointx += ImageSizeX;
                        Point1Proj += PixInterval;
                    } else {
                        float coef = (DetCol1Proj - Bound0) / PixArea;
                        if (FBPWEIGHT) coef /= CoordinateWeight(sourcex, sourcey, Pointx, Pointy);
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            float temp = (Point1Proj2z - Bound0z) * coef * ProjTemp[idxd];
                            atomicAdd(Image + ((idxBatch * Depth + idxslice) * Height + idxrow) * Width + idxcol, temp);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = DetCol1Proj;
                        idxd ++;
                        if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                    }
                }
            }
        }
    } else {
        if (idxDetCol < NumDetCol) {
            int idxDetColTemp = (sinVal >= 0) ? idxDetCol : (NumDetCol - 1 - idxDetCol);
            ProjTemp[tx] = Projection[(blockIdx.x * NumDetRow + idxDetRow) * NumDetCol + idxDetColTemp] * DetColRad;
        } else {
            ProjTemp[tx] = 0;
        }
        float DetColx;
        if (sinVal >= 0) {
            DetCol0Ang = (idxDetCol0 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (idxDetCol + 1 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColx = sin(DetCol1Ang - 0.5 * DetColSize) * SourceDetector + sourcex;
            flag = 1;
        } else {
            DetCol0Ang = (NumDetCol / 2.0 - idxDetCol0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (NumDetCol / 2.0 - idxDetCol - 1) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColx = sin(DetCol1Ang + 0.5 * DetColSize) * SourceDetector + sourcex;
            flag = - 1;
        }
        float DetCol0Proj = DetMap2y(sourcex, sourcey, DetCol0Ang);
        float DetCol1Proj = DetMap2y(sourcex, sourcey, DetCol1Ang);
        float DetRow0Proj = sourcex / (sourcex - DetColx) * DetRow0z;
        float DetRow1Proj = sourcex / (sourcex - DetColx) * DetRow1z;
        if (tx == 0) DetColProj2Axis[tx] = DetCol0Proj;
        DetColProj2Axis[tx + 1] = DetCol1Proj;
        DetRow0Proj2Axis[tx] = DetRow0Proj;
        DetRow1Proj2Axis[tx] = DetRow1Proj;
        __syncthreads();
        float Point0y = - Height / 2.0 * ImageSizeY + PixYShift;
        float Point1y = Height / 2.0 * ImageSizeY + PixYShift;
        float Point0z = - Depth / 2.0 * ImageSizeZ + PixZShift;
        float Point1z = Depth / 2.0 * ImageSizeZ + PixZShift;
        for (int i = 0; (i * blockDim.x) < Width; i++) {
            int idxcol = i * blockDim.x + tx;
            if (idxcol < Width) {
                float Pointx = (idxcol - Width / 2.0 + 0.5) * ImageSizeX + PixXShift;
                float Point0Proj = Map2y(sourcex, sourcey, Pointx, Point0y);
                float Point1Proj = Map2y(sourcex, sourcey, Pointx, Point1y);
                float Point0Proj2z = sourcex / (sourcex - Pointx) * Point0z;
                float Point1Proj2z = sourcex / (sourcex - Pointx) * Point1z;
                float PixInterval = (Point1Proj - Point0Proj) / Height;
                float PixIntervalz = (Point1Proj2z - Point0Proj2z) / Depth;
                float PixArea = PixInterval * PixIntervalz;
                float tanVal0 = (sourcey - Point0Proj) / sourcex;
                float tanVal1 = - 1 / tan(BinColShift + ViewAngle[idxView]);
                float delta = atan((tanVal0 - tanVal1) / (1 + tanVal0 * tanVal1));
                int idxd = floor(NumDetCol / 2.0 - idxDetCol0 + delta * flag / DetColSize);
                int idxrow;
                float Bound0;
                if (idxd < 0) {
                    Bound0 = DetColProj2Axis[0];
                    idxrow = floor((Bound0 - Point0Proj) / PixInterval);
                    idxd = 0;
                } else {
                    Bound0 = Point0Proj;
                    idxrow = 0;
                }
                float Pointy = (idxrow + 0.5) * ImageSizeY + Point0y;
                Point1Proj = (idxrow + 1) * PixInterval + Point0Proj;
                if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                while(idxrow < Height && idxd < MaxNDetCol) {
                    float Bound0z = max(Point0Proj2z, DetRow0Proj2Axis[idxd]);
                    float Bound1z = min(Point1Proj2z, DetRow1Proj2Axis[idxd]);
                    int idxslice;
                    if (Bound0z == Point0Proj2z) {
                        idxslice = 0;
                    } else {
                        idxslice = floor((Bound0z - Point0Proj2z) / PixIntervalz);
                    }
                    float Point1Proj2z = (idxslice + 1) * PixIntervalz + Point0Proj2z;
                    if (Point1Proj < DetCol1Proj) {
                        float coef = (Point1Proj - Bound0) / PixArea;
                        if (FBPWEIGHT) coef /= CoordinateWeight(sourcex, sourcey, Pointx, Pointy);
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            float temp = (Point1Proj2z - Bound0z) * coef * ProjTemp[idxd];
                            atomicAdd(Image + ((idxBatch * Depth + idxslice) * Height + Height - 1 - idxrow) * Width + idxcol, temp);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = Point1Proj;
                        idxrow++;
                        Pointy += ImageSizeY;
                        Point1Proj += PixInterval;
                    } else {
                        float coef = (DetCol1Proj - Bound0) / PixArea;
                        if (FBPWEIGHT) coef /= CoordinateWeight(sourcex, sourcey, Pointx, Pointy);
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            float temp = (Point1Proj2z - Bound0z) * coef * ProjTemp[idxd];
                            atomicAdd(Image + ((idxBatch * Depth + idxslice) * Height + Height - 1 - idxrow) * Width + idxcol, temp);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = DetCol1Proj;
                        idxd ++;
                        if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                    }
                }
            }
        }
    }
}


template<bool FBPWEIGHT>
__global__ void BackProjTransConeArcDisCUDA3dKernel(
    float* __restrict__ Projection,
    hipTextureObject_t texObj,
    const float* __restrict__ ViewAngle,
    const int Width,
    const int Height,
    const int Depth,
    const int NumView,
    const int NumDetCol,
    const int NumDetRow,
    const float ImageSizeX,
    const float ImageSizeY,
    const float ImageSizeZ,
    const float DetColSize,
    const float DetRowSize,
    const float IsoSource,
    const float SourceDetector,
    const float PixXShift,
    const float PixYShift,
    const float PixZShift,
    const float BinColShift,
    const float BinRowShift) {

    const int idxBatch = blockIdx.x / NumView;
    const int idxView = blockIdx.x % NumView;
    const int idxDetRow = blockIdx.y;
    const int idxDetCol0 = blockIdx.z * blockDim.x;
    const int tx = threadIdx.x;
    const int idxDetCol = idxDetCol0 + tx;
    const int MaxNDetCol = ((idxDetCol0 + blockDim.x) > NumDetCol) ? (NumDetCol - idxDetCol0) : blockDim.x;
    __shared__ float ProjTemp[BLOCK_DIM];
    __shared__ float DetRow0Proj2Axis[BLOCK_DIM];
    __shared__ float DetRow1Proj2Axis[BLOCK_DIM];
    __shared__ float DetColProj2Axis[BLOCK_DIM + 1];

    float sinVal = sin(ViewAngle[idxView]);
    float cosVal = cos(ViewAngle[idxView]);
    float sourcex = - sinVal * IsoSource;
    float sourcey = cosVal * IsoSource;
    float DetColRad = IsoSource * DetColSize;
    float DetRow0z = (idxDetRow - NumDetRow / 2.0) * DetRowSize + BinRowShift;
    float DetRow1z = DetRow0z + DetRowSize;
    float DetCol0Ang;
    float DetCol1Ang;
    int flag;
    if (cosVal * cosVal > 0.5) {
        ProjTemp[tx] = 0;
        float DetColy;
        if (cosVal >= 0) {
            DetCol0Ang = (idxDetCol0 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (idxDetCol + 1 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColy = - cos(DetCol1Ang - 0.5 * DetColSize) * SourceDetector + sourcey;
            flag = 1;
        } else {
            DetCol0Ang = (NumDetCol / 2.0 - idxDetCol0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (NumDetCol / 2.0 - idxDetCol - 1) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColy = - cos(DetCol1Ang + 0.5 * DetColSize) * SourceDetector + sourcey;
            flag = - 1;
        }
        float DetCol0Proj = DetMap2x(sourcex, sourcey, DetCol0Ang);
        float DetCol1Proj = DetMap2x(sourcex, sourcey, DetCol1Ang);
        float DetRow0Proj = sourcey / (sourcey - DetColy) * DetRow0z;
        float DetRow1Proj = sourcey / (sourcey - DetColy) * DetRow1z;
        if (tx == 0) DetColProj2Axis[tx] = DetCol0Proj;
        DetColProj2Axis[tx + 1] = DetCol1Proj;
        DetRow0Proj2Axis[tx] = DetRow0Proj;
        DetRow1Proj2Axis[tx] = DetRow1Proj;
        __syncthreads();
        float Point0x = - Width / 2.0 * ImageSizeX + PixXShift;
        float Point1x = Width / 2.0 * ImageSizeX + PixXShift;
        float Point0z = - Depth / 2.0 * ImageSizeZ + PixZShift;
        float Point1z = Depth / 2.0 * ImageSizeZ + PixZShift;
        for (int i = 0; (i * blockDim.x) < Height; i++) {
            int idxrow = i * blockDim.x + tx;
            if (idxrow < Height) {
                float Pointy = (Height / 2.0 - idxrow - 0.5) * ImageSizeY + PixYShift;
                float Point0Proj = Map2x(sourcex, sourcey, Point0x, Pointy);
                float Point1Proj = Map2x(sourcex, sourcey, Point1x, Pointy);
                float Point0Proj2z = sourcey / (sourcey - Pointy) * Point0z;
                float Point1Proj2z = sourcey / (sourcey - Pointy) * Point1z;
                float PixInterval = (Point1Proj - Point0Proj) / Width;
                float PixIntervalz = (Point1Proj2z - Point0Proj2z) / Depth;
                float PixArea = PixInterval * PixIntervalz;
                float tanVal0 = ((sourcex - Point0Proj) == 0)? 1e10 : sourcey / (sourcex - Point0Proj);
                float tanVal1 = ((BinColShift + ViewAngle[idxView]) == 0)? 1e10 : - 1 / tan(BinColShift + ViewAngle[idxView]);
                float delta = atan((tanVal0 - tanVal1) / (1 + tanVal0 * tanVal1));
                int idxd = floor(NumDetCol / 2.0 - idxDetCol0 + delta * flag / DetColSize);
                int idxcol;
                float Bound0;
                if (idxd < 0) {
                    Bound0 = DetColProj2Axis[0];
                    idxcol = floor((Bound0 - Point0Proj) / PixInterval);
                    idxd = 0;
                } else {
                    Bound0 = Point0Proj;
                    idxcol = 0;
                }
                float Pointx = (idxcol + 0.5) * ImageSizeX + Point0x;
                Point1Proj = (idxcol + 1) * PixInterval + Point0Proj;
                if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                float temp = 0;
                while(idxcol < Width && idxd < MaxNDetCol) {
                    float Bound0z = max(Point0Proj2z, DetRow0Proj2Axis[idxd]);
                    float Bound1z = min(Point1Proj2z, DetRow1Proj2Axis[idxd]);
                    int idxslice;
                    if (Bound0z == Point0Proj2z) {
                        idxslice = 0;
                    } else {
                        idxslice = floor((Bound0z - Point0Proj2z) / PixIntervalz);
                    }
                    float Point1Proj2z = (idxslice + 1) * PixIntervalz + Point0Proj2z;
                    if (Point1Proj < DetCol1Proj) {
                        float coef = (Point1Proj - Bound0) / PixArea;
                        if (FBPWEIGHT) coef /= CoordinateWeight(sourcex, sourcey, Pointx, Pointy);
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            temp += (Point1Proj2z - Bound0z) * coef * tex3D<float>(texObj, idxcol, idxrow, idxBatch * Depth + idxslice);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = Point1Proj;
                        idxcol++;
                        Pointx += ImageSizeX;
                        Point1Proj += PixInterval;
                    } else {
                        float coef = (DetCol1Proj - Bound0) / PixArea;
                        if (FBPWEIGHT) coef /= CoordinateWeight(sourcex, sourcey, Pointx, Pointy);
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            temp += (Point1Proj2z - Bound0z) * coef * tex3D<float>(texObj, idxcol, idxrow, idxBatch * Depth + idxslice);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        atomicAdd(ProjTemp + idxd, temp);
                        temp = 0;
                        Bound0 = DetCol1Proj;
                        idxd ++;
                        if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                    }
                }
                if (temp != 0) atomicAdd(ProjTemp + idxd, temp);
            }
        }
        __syncthreads();
        if (idxDetCol < NumDetCol) {
            ProjTemp[tx] *= DetColRad;
            int idxDetColTemp = (cosVal >= 0) ? idxDetCol : (NumDetCol - 1 - idxDetCol);
            Projection[(blockIdx.x * NumDetRow + idxDetRow) * NumDetCol + idxDetColTemp] = ProjTemp[tx];
        }
    } else {
        ProjTemp[tx] = 0;
        float DetColx;
        if (sinVal >= 0) {
            DetCol0Ang = (idxDetCol0 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (idxDetCol + 1 - NumDetCol / 2.0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColx = sin(DetCol1Ang - 0.5 * DetColSize) * SourceDetector + sourcex;
            flag = 1;
        } else {
            DetCol0Ang = (NumDetCol / 2.0 - idxDetCol0) * DetColSize + BinColShift + ViewAngle[idxView];
            DetCol1Ang = (NumDetCol / 2.0 - idxDetCol - 1) * DetColSize + BinColShift + ViewAngle[idxView];
            DetColx = sin(DetCol1Ang + 0.5 * DetColSize) * SourceDetector + sourcex;
            flag = - 1;
        }
        float DetCol0Proj = DetMap2y(sourcex, sourcey, DetCol0Ang);
        float DetCol1Proj = DetMap2y(sourcex, sourcey, DetCol1Ang);
        float DetRow0Proj = sourcex / (sourcex - DetColx) * DetRow0z;
        float DetRow1Proj = sourcex / (sourcex - DetColx) * DetRow1z;
        if (tx == 0) DetColProj2Axis[tx] = DetCol0Proj;
        DetColProj2Axis[tx + 1] = DetCol1Proj;
        DetRow0Proj2Axis[tx] = DetRow0Proj;
        DetRow1Proj2Axis[tx] = DetRow1Proj;
        __syncthreads();
        float Point0y = - Height / 2.0 * ImageSizeY + PixYShift;
        float Point1y = Height / 2.0 * ImageSizeY + PixYShift;
        float Point0z = - Depth / 2.0 * ImageSizeZ + PixZShift;
        float Point1z = Depth / 2.0 * ImageSizeZ + PixZShift;
        for (int i = 0; (i * blockDim.x) < Width; i++) {
            int idxcol = i * blockDim.x + tx;
            if (idxcol < Width) {
                float Pointx = (idxcol - Width / 2.0 + 0.5) * ImageSizeX + PixXShift;
                float Point0Proj = Map2y(sourcex, sourcey, Pointx, Point0y);
                float Point1Proj = Map2y(sourcex, sourcey, Pointx, Point1y);
                float Point0Proj2z = sourcex / (sourcex - Pointx) * Point0z;
                float Point1Proj2z = sourcex / (sourcex - Pointx) * Point1z;
                float PixInterval = (Point1Proj - Point0Proj) / Height;
                float PixIntervalz = (Point1Proj2z - Point0Proj2z) / Depth;
                float PixArea = PixInterval * PixIntervalz;
                float tanVal0 = (sourcey - Point0Proj) / sourcex;
                float tanVal1 = - 1 / tan(BinColShift + ViewAngle[idxView]);
                float delta = atan((tanVal0 - tanVal1) / (1 + tanVal0 * tanVal1));
                int idxd = floor(NumDetCol / 2.0 - idxDetCol0 + delta * flag / DetColSize);
                int idxrow;
                float Bound0;
                if (idxd < 0) {
                    Bound0 = DetColProj2Axis[0];
                    idxrow = floor((Bound0 - Point0Proj) / PixInterval);
                    idxd = 0;
                } else {
                    Bound0 = Point0Proj;
                    idxrow = 0;
                }
                float Pointy = (idxrow + 0.5) * ImageSizeY + Point0y;
                Point1Proj = (idxrow + 1) * PixInterval + Point0Proj;
                if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                float temp = 0;
                while(idxrow < Height && idxd < MaxNDetCol) {
                    float Bound0z = max(Point0Proj2z, DetRow0Proj2Axis[idxd]);
                    float Bound1z = min(Point1Proj2z, DetRow1Proj2Axis[idxd]);
                    int idxslice;
                    if (Bound0z == Point0Proj2z) {
                        idxslice = 0;
                    } else {
                        idxslice = floor((Bound0z - Point0Proj2z) / PixIntervalz);
                    }
                    float Point1Proj2z = (idxslice + 1) * PixIntervalz + Point0Proj2z;
                    if (Point1Proj < DetCol1Proj) {
                        float coef = (Point1Proj - Bound0) / PixArea;
                        if (FBPWEIGHT) coef /= CoordinateWeight(sourcex, sourcey, Pointx, Pointy);
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            temp += (Point1Proj2z - Bound0z) * coef * tex3D<float>(texObj, idxcol, Height - 1 - idxrow, idxBatch * Depth + idxslice);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        Bound0 = Point1Proj;
                        idxrow++;
                        Pointy += ImageSizeY;
                        Point1Proj += PixInterval;
                    } else {
                        float coef = (DetCol1Proj - Bound0) / PixArea;
                        if (FBPWEIGHT) coef /= CoordinateWeight(sourcex, sourcey, Pointx, Pointy);
                        while (Bound0z < Bound1z && idxslice < Depth) {
                            Point1Proj2z = (Point1Proj2z > Bound1z) ? Bound1z : Point1Proj2z;
                            temp += (Point1Proj2z - Bound0z) * coef * tex3D<float>(texObj, idxcol, Height - 1 - idxrow, idxBatch * Depth + idxslice);
                            Bound0z = Point1Proj2z;
                            idxslice++;
                            Point1Proj2z += PixIntervalz;
                        }
                        atomicAdd(ProjTemp + idxd, temp);
                        temp = 0;
                        Bound0 = DetCol1Proj;
                        idxd ++;
                        if (idxd < MaxNDetCol) DetCol1Proj = DetColProj2Axis[idxd + 1];
                    }
                }
                if (temp != 0) atomicAdd(ProjTemp + idxd, temp);
            }
        }
        __syncthreads();
        if (idxDetCol < NumDetCol) {
            ProjTemp[tx] *= DetColRad;
            int idxDetColTemp = (sinVal >= 0) ? idxDetCol : (NumDetCol - 1 - idxDetCol);
            Projection[(blockIdx.x * NumDetRow + idxDetRow) * NumDetCol + idxDetColTemp] = ProjTemp[tx];
        }
    }
}


void ProjConeArcDisCUDA3d(
    float *Image,
    float *Projection,
    float *ViewAngle,
    int BatchSize,
    int Width,
    int Height,
    int Depth,
    int NumView,
    int NumDetCol,
    int NumDetRow,
    float ImageSizeX,
    float ImageSizeY,
    float ImageSizeZ,
    float DetColSize,
    float DetRowSize,
    float IsoSource,
    float SourceDetector,
    float PixXShift,
    float PixYShift,
    float PixZShift,
    float BinColShift,
    float BinRowShift) {

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    hipExtent extent = make_hipExtent(Width, Height, BatchSize * Depth);
    hipMalloc3DArray(&cuArray, &channelDesc, extent);

    hipMemcpy3DParms copyParams={0};
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = make_hipPitchedPtr((void *)Image, extent.width * sizeof(float), extent.width, extent.height);
    copyParams.dstArray = cuArray;
    hipMemcpy3D(&copyParams);

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.addressMode[2] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    int NumBlockZ = (NumDetCol - 1) / BLOCK_DIM + 1;
    const dim3 GridSize(BatchSize * NumView, NumDetRow, NumBlockZ);

    ProjConeArcDisCUDA3dKernel<<<GridSize, BLOCK_DIM>>>(
        Projection, texObj, ViewAngle, Width, Height, Depth, NumView, NumDetCol, NumDetRow, ImageSizeX, ImageSizeY, ImageSizeZ,
        DetColSize, DetRowSize, IsoSource, SourceDetector, PixXShift, PixYShift, PixZShift, BinColShift, BinRowShift
    );

    hipDestroyTextureObject(texObj);
    hipFreeArray(cuArray);
}


void ProjTransConeArcDisCUDA3d(
    float *Image,
    float *Projection,
    float *ViewAngle,
    int BatchSize,
    int Width,
    int Height,
    int Depth,
    int NumView,
    int NumDetCol,
    int NumDetRow,
    float ImageSizeX,
    float ImageSizeY,
    float ImageSizeZ,
    float DetColSize,
    float DetRowSize,
    float IsoSource,
    float SourceDetector,
    float PixXShift,
    float PixYShift,
    float PixZShift,
    float BinColShift,
    float BinRowShift) {

    int NumBlockZ = (NumDetCol - 1) / BLOCK_DIM + 1;
    const dim3 GridSize(BatchSize * NumView, NumDetRow, NumBlockZ);

    ProjTransConeArcDisCUDA3dKernel<<<GridSize, BLOCK_DIM>>>(
        Image, Projection, ViewAngle, Width, Height, Depth, NumView, NumDetCol, NumDetRow, ImageSizeX, ImageSizeY, ImageSizeZ,
        DetColSize, DetRowSize, IsoSource, SourceDetector, PixXShift, PixYShift, PixZShift, BinColShift, BinRowShift
    );
}


void BackProjConeArcDisCUDA3d(
    float *Image,
    float *Projection,
    float *ViewAngle,
    int BatchSize,
    int Width,
    int Height,
    int Depth,
    int NumView,
    int NumDetCol,
    int NumDetRow,
    float ImageSizeX,
    float ImageSizeY,
    float ImageSizeZ,
    float DetColSize,
    float DetRowSize,
    float IsoSource,
    float SourceDetector,
    float PixXShift,
    float PixYShift,
    float PixZShift,
    float BinColShift,
    float BinRowShift) {

    int NumBlockZ = (NumDetCol - 1) / BLOCK_DIM + 1;
    const dim3 GridSize(BatchSize * NumView, NumDetRow, NumBlockZ);

    BackProjConeArcDisCUDA3dKernel<false><<<GridSize, BLOCK_DIM>>>(
        Image, Projection, ViewAngle, Width, Height, Depth, NumView, NumDetCol, NumDetRow, ImageSizeX, ImageSizeY, ImageSizeZ,
        DetColSize, DetRowSize, IsoSource, SourceDetector, PixXShift, PixYShift, PixZShift, BinColShift, BinRowShift
    );
}


void BackProjTransConeArcDisCUDA3d(
    float *Image,
    float *Projection,
    float *ViewAngle,
    int BatchSize,
    int Width,
    int Height,
    int Depth,
    int NumView,
    int NumDetCol,
    int NumDetRow,
    float ImageSizeX,
    float ImageSizeY,
    float ImageSizeZ,
    float DetColSize,
    float DetRowSize,
    float IsoSource,
    float SourceDetector,
    float PixXShift,
    float PixYShift,
    float PixZShift,
    float BinColShift,
    float BinRowShift) {

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    hipExtent extent = make_hipExtent(Width, Height, BatchSize * Depth);
    hipMalloc3DArray(&cuArray, &channelDesc, extent);

    hipMemcpy3DParms copyParams={0};
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = make_hipPitchedPtr((void *)Image, extent.width * sizeof(float), extent.width, extent.height);
    copyParams.dstArray = cuArray;
    hipMemcpy3D(&copyParams);

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.addressMode[2] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    int NumBlockZ = (NumDetCol - 1) / BLOCK_DIM + 1;
    const dim3 GridSize(BatchSize * NumView, NumDetRow, NumBlockZ);

    BackProjTransConeArcDisCUDA3dKernel<false><<<GridSize, BLOCK_DIM>>>(
        Projection, texObj, ViewAngle, Width, Height, Depth, NumView, NumDetCol, NumDetRow, ImageSizeX, ImageSizeY, ImageSizeZ,
        DetColSize, DetRowSize, IsoSource, SourceDetector, PixXShift, PixYShift, PixZShift, BinColShift, BinRowShift
    );

    hipDestroyTextureObject(texObj);
    hipFreeArray(cuArray);
}


void BackProjWeightedConeArcDisCUDA3d(
    float *Image,
    float *Projection,
    float *ViewAngle,
    int BatchSize,
    int Width,
    int Height,
    int Depth,
    int NumView,
    int NumDetCol,
    int NumDetRow,
    float ImageSizeX,
    float ImageSizeY,
    float ImageSizeZ,
    float DetColSize,
    float DetRowSize,
    float IsoSource,
    float SourceDetector,
    float PixXShift,
    float PixYShift,
    float PixZShift,
    float BinColShift,
    float BinRowShift) {

    int NumBlockZ = (NumDetCol - 1) / BLOCK_DIM + 1;
    const dim3 GridSize(BatchSize * NumView, NumDetRow, NumBlockZ);

    BackProjConeArcDisCUDA3dKernel<true><<<GridSize, BLOCK_DIM>>>(
        Image, Projection, ViewAngle, Width, Height, Depth, NumView, NumDetCol, NumDetRow, ImageSizeX, ImageSizeY, ImageSizeZ,
        DetColSize, DetRowSize, IsoSource, SourceDetector, PixXShift, PixYShift, PixZShift, BinColShift, BinRowShift
    );
}


void BackProjTransWeightedConeArcDisCUDA3d(
    float *Image,
    float *Projection,
    float *ViewAngle,
    int BatchSize,
    int Width,
    int Height,
    int Depth,
    int NumView,
    int NumDetCol,
    int NumDetRow,
    float ImageSizeX,
    float ImageSizeY,
    float ImageSizeZ,
    float DetColSize,
    float DetRowSize,
    float IsoSource,
    float SourceDetector,
    float PixXShift,
    float PixYShift,
    float PixZShift,
    float BinColShift,
    float BinRowShift) {

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    hipExtent extent = make_hipExtent(Width, Height, BatchSize * Depth);
    hipMalloc3DArray(&cuArray, &channelDesc, extent);

    hipMemcpy3DParms copyParams={0};
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = make_hipPitchedPtr((void *)Image, extent.width * sizeof(float), extent.width, extent.height);
    copyParams.dstArray = cuArray;
    hipMemcpy3D(&copyParams);

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.addressMode[2] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    int NumBlockZ = (NumDetCol - 1) / BLOCK_DIM + 1;
    const dim3 GridSize(BatchSize * NumView, NumDetRow, NumBlockZ);

    BackProjTransConeArcDisCUDA3dKernel<true><<<GridSize, BLOCK_DIM>>>(
        Projection, texObj, ViewAngle, Width, Height, Depth, NumView, NumDetCol, NumDetRow, ImageSizeX, ImageSizeY, ImageSizeZ,
        DetColSize, DetRowSize, IsoSource, SourceDetector, PixXShift, PixYShift, PixZShift, BinColShift, BinRowShift
    );

    hipDestroyTextureObject(texObj);
    hipFreeArray(cuArray);
}